#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>


__global__ void kernel(int *dev_array, int N){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if(tId < N){
        dev_array[tId] = 1;
    }
}




/*
 *  Procesamiento Imagen CPU
 */
void funcionCPU(){

}

/*
 *  Procesamiento Imagen GPU
 */
__global__ void kernelGPU(){

}

/*
 *  Codigo Principal
 */
int main(int argc, char **argv){

    // /*
    //  *  Inicializacion
    //  */
	// clock_t t1, t2;
	// cudaEvent_t ct1, ct2;
	// double ms;
	// float dt;
	// int M, N;
    // float *Rhost, *Ghost, *Bhost;
    // float *Rhostout, *Ghostout, *Bhostout;
    // float *Rdev, *Gdev, *Bdev;
    // float *Rdevout, *Gdevout, *Bdevout;

    // Read(&Rhost, &Ghost, &Bhost, &M, &N, "img.txt");

    // /*
    //  *  Parte CPU
    //  */
    // Rhostout = new float[M*N];
    // Ghostout = new float[M*N];
    // Bhostout = new float[M*N];

    // t1 = clock();
    // funcion(); // Agregar parametros!
    // t2 = clock();
    // ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
    // std::cout << "Tiempo CPU: " << ms << "[ms]" << std::endl;
    // Write(Rhostout, Ghostout, Bhostout, M, N, "imgCPU.txt");

    // delete[] Rhostout; delete[] Ghostout; delete[] Bhostout;
    
    // /*
    //  *  Parte GPU
    //  */

    // int grid_size, block_size = 256;
    // grid_size = (int)ceil((float) M * N / block_size);
        
    // cudaMalloc((void**)&Rdev, M * N * sizeof(float));
    // cudaMalloc((void**)&Gdev, M * N * sizeof(float));
    // cudaMalloc((void**)&Bdev, M * N * sizeof(float));
    // cudaMemcpy(Rdev, Rhost, M * N * sizeof(float), cudaMemcpyHostToDevice);
    // cudaMemcpy(Gdev, Ghost, M * N * sizeof(float), cudaMemcpyHostToDevice);
    // cudaMemcpy(Bdev, Bhost, M * N * sizeof(float), cudaMemcpyHostToDevice);
        
    // cudaMalloc((void**)&Rdevout, M * N * sizeof(float));
    // cudaMalloc((void**)&Gdevout, M * N * sizeof(float));
    // cudaMalloc((void**)&Bdevout, M * N * sizeof(float));
    
    // cudaEventCreate(&ct1);
    // cudaEventCreate(&ct2);
    // cudaEventRecord(ct1);
    // kernel<<<grid_size, block_size>>>(); // Agregar parametros!
    // cudaEventRecord(ct2);
    // cudaEventSynchronize(ct2);
    // cudaEventElapsedTime(&dt, ct1, ct2);
    // std::cout << "Tiempo GPU: " << dt << "[ms]" << std::endl;

    // Rhostout = new float[M*N];
    // Ghostout = new float[M*N];
    // Bhostout = new float[M*N];
    // cudaMemcpy(Rhostout, Rdevout, M * N * sizeof(float), cudaMemcpyDeviceToHost);
    // cudaMemcpy(Ghostout, Gdevout, M * N * sizeof(float), cudaMemcpyDeviceToHost);
    // cudaMemcpy(Bhostout, Bdevout, M * N * sizeof(float), cudaMemcpyDeviceToHost);
    // Write(Rhostout, Ghostout, Bhostout, M, N, "imgGPU.txt");

    // cudaFree(Rdev); cudaFree(Gdev); cudaFree(Bdev);
    // cudaFree(Rdevout); cudaFree(Gdevout); cudaFree(Bdevout);
    // delete[] Rhost; delete[] Ghost; delete[] Bhost;
    // delete[] Rhostout; delete[] Ghostout; delete[] Bhostout;
    
    int N = 100;
    int block_size = 256;
    int grid_size = (int)ceil((float)N / block_size);
    // std::cout << 0 << "\n";
    int *host_array = new int[N];
    int *dev_array;
    hipMalloc(&dev_array, N* sizeof(int));
    kernel<<<grid_size, block_size>>>(dev_array, N);
    hipMemcpy(host_array, dev_array, N* sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_array);
    std::cout << "(";
    for (int i = 0; i < N; i++){
        std::cout << host_array[i] << " ";
    }
    std::cout << ")\n";
	return 0;
}